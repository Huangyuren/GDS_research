#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define BLOCK_SIZE 1024
inline hipError_t cudaCheckError(hipError_t result) {
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA error %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

__global__ void matrixMultiplication() {}


int main(int argc, char* argv[]) {
	int row_a = atoi(argv[1]);
	int col_a = atoi(argv[2]);
	int col_b = atoi(argv[3]);
	int* h_a, *h_b, *h_c;
	cudaCheckError(hipHostMalloc(&h_a, sizeof(int)*(row_a*col_a)));
	cudaCheckError(hipHostMalloc(&h_b, sizeof(int)*(col_a*col_b)));
	cudaCheckError(hipHostMalloc(&h_c, sizeof(int)*(row_a*col_b)));
	
	int* dev_a, *dev_b, *dev_c;
	cudaCheckError(hipMalloc(&dev_a, sizeof(int)*(row_a*col_a)));
	cudaCheckError(hipMalloc(&dev_b, sizeof(int)*(col-a*col_b)));
	cudaCheckError(hipMalloc(&dev_c, sizeof(int)*(row_a*col_b)));

	cudaCheckError(hipMemcpy(dev_a, h_a, sizeof(int)*(row_a*col_a), hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(dev_b, h_b, sizeof(int)*(col_a*col_b), hipMemcpyHostToDevice));

    int grid_row = (row_a + BLOCK_SIZE +1) / BLOCK_SIZE;
    int grid_col = (col_b + BLOCK_SIZE +1) / BLOCK_SIZE;
	dim3 dimGrid(grid_col, grid_row);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	matrixMultiplication<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c);
}
